#include "hip/hip_runtime.h"
#include "../include/CMaT.h"

int main (  ) {

    // Specify AMGX configuration.
    std::string config_spec = "config_version=2, \
                               solver(main)=FGMRES, \
                               main:max_iters=300, \
                               main:convergence=RELATIVE_MAX, \
                               main:tolerance=0.00000001, \
                               main:monitor_residual=1, \
                               main:preconditioner(amg)=AMG, \
                               main:print_solve_stats=1, \
                               amg:algorithm=AGGREGATION, \
                               amg:selector=SIZE_8, \
                               amg:cycle=V, \
                               amg:max_iters=1, \
                               amg:max_levels=10, \
                               amg:smoother(amg_smoother)=BLOCK_JACOBI, \
                               amg:relaxation_factor=0.75, \
                               amg:presweeps=1, \
                               amg:postsweeps=2, \
                               amg:coarsest_sweeps=4, \
                               determinism_flag=1";

    // Open the MAT file.
    matfile_load mat_file_load("./tests/MAT_FILES","Poisson_1000000.mat");

    // Load the data (Ar=b).
    GPU_Sparse A(mat_file_load,"A");
    GPU_Dense  b(mat_file_load,"b");
    GPU_Dense  r(mat_file_load,"x");

    // Solve Ax=b.
    tic();
    GPU_Dense x = Solve(A,b,config_spec);
    toc();

    // Subtract the r from x.
    GPU_Dense b_x = A.mv(x);
    GPU_Dense res = b_x.add(b.scp(-1));

    // Print the result.
    std::cout << "2-norm of the residual is: " << sqrt(res.dot(res)) << std::endl;

    return 0;

}