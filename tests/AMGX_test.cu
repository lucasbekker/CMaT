#include "hip/hip_runtime.h"
#include "../include/CMaT.h"

int CPU_AMGX_test ( std::string config_spec ) {

    // Check.
    int pass = 1;
    double tol = 0.0001;

    // Initialize problem data.
    CPU_Sparse A(4,4,10);
    CPU_Dense  b(1,4);
    CPU_Dense  x_0(1,4);

    // Fill A with data.
    A.Values[0] = 2; A.Values[1] = 1; A.Values[2] = 1;
    A.Values[3] = 2; A.Values[4] = 1; A.Values[5] = 1;
    A.Values[6] = 2; A.Values[7] = 1; A.Values[8] = 1;
    A.Values[9] = 2;
    A.J[0] = 0; A.J[1] = 1; A.J[2] = 0; A.J[3] = 1; A.J[4] = 2;
    A.J[5] = 1; A.J[6] = 2; A.J[7] = 3; A.J[8] = 2; A.J[9] = 3;
    A.Ib[0] = 0; A.Ib[1] = 2; A.Ib[2] = 5; A.Ib[3] = 8;
    A.Ie[0] = 2; A.Ie[1] = 5; A.Ie[2] = 8; A.Ie[3] = 10;
    
    // Fill b with data.
    b.Values[0] = 1; b.Values[1] = 1; b.Values[2] = 1;
    b.Values[3] = 1;

    // Fill x with data. (initial guess)
    x_0.Values[0] = 1; x_0.Values[1] = 1; x_0.Values[2] = 1;
    x_0.Values[3] = 1;

    // Copy x_0 to x;
    CPU_Dense x = x_0.clone();

    // Initialize solver data and fill with addresses.
    SOLVER_data Axb;
    Axb.A_c = &A;
    Axb.b_c = &b;
    Axb.x_c = &x;
    Axb.n = A.Size[0];
    Axb.nnz = A.Size[2];

    // Specify AMGX configuration.
    AMGX_Mode mode = AMGX_mode_hDDI;
    
    // Start the solving procedure.
    SOLVER_AmgX AMGX(config_spec, mode, Axb);

    // Check for errors.
    if ( abs(x.Values[0] - 0.4) > tol ||
         abs(x.Values[1] - 0.2) > tol ||
         abs(x.Values[2] - 0.2) > tol ||
         abs(x.Values[3] - 0.4) > tol ) {
         pass = 0; }

    // Return the result.
    return pass;

}

int CPU_f_AMGX_test ( std::string config_spec ) {

    // Check.
    int pass = 1;
    float tol = 0.0001;

    // Initialize problem data.
    CPU_Sparse_f A(4,4,10);
    CPU_Dense_f  b(1,4);
    CPU_Dense_f  x_0(1,4);

    // Fill A with data.
    A.Values[0] = 2; A.Values[1] = 1; A.Values[2] = 1;
    A.Values[3] = 2; A.Values[4] = 1; A.Values[5] = 1;
    A.Values[6] = 2; A.Values[7] = 1; A.Values[8] = 1;
    A.Values[9] = 2;
    A.J[0] = 0; A.J[1] = 1; A.J[2] = 0; A.J[3] = 1; A.J[4] = 2;
    A.J[5] = 1; A.J[6] = 2; A.J[7] = 3; A.J[8] = 2; A.J[9] = 3;
    A.Ib[0] = 0; A.Ib[1] = 2; A.Ib[2] = 5; A.Ib[3] = 8;
    A.Ie[0] = 2; A.Ie[1] = 5; A.Ie[2] = 8; A.Ie[3] = 10;
    
    // Fill b with data.
    b.Values[0] = 1; b.Values[1] = 1; b.Values[2] = 1;
    b.Values[3] = 1;

    // Fill x with data. (initial guess)
    x_0.Values[0] = 1; x_0.Values[1] = 1; x_0.Values[2] = 1;
    x_0.Values[3] = 1;

    // Copy x_0 to x;
    CPU_Dense_f x = x_0.clone();

    // Initialize solver data and fill with addresses.
    SOLVER_data Axb;
    Axb.A_cf = &A;
    Axb.b_cf = &b;
    Axb.x_cf = &x;
    Axb.n = A.Size[0];
    Axb.nnz = A.Size[2];

    // Specify AMGX configuration.
    AMGX_Mode mode = AMGX_mode_hFFI;
    
    // Start the solving procedure.
    SOLVER_AmgX AMGX(config_spec, mode, Axb);

    // Check for errors.
    if ( abs(x.Values[0] - 0.4) > tol ||
         abs(x.Values[1] - 0.2) > tol ||
         abs(x.Values[2] - 0.2) > tol ||
         abs(x.Values[3] - 0.4) > tol ) {
         pass = 0; }

    // Return the result.
    return pass;

}

int GPU_AMGX_test ( std::string config_spec ) {

    // Check.
    int pass = 1;
    double tol = 0.0001;

    // Initialize problem data.
    GPU_Sparse A(4,4,10);
    GPU_Dense  b(1,4);
    GPU_Dense  x_0(1,4);

    // Fill A with data.
    A.Values[0] = 2; A.Values[1] = 1; A.Values[2] = 1;
    A.Values[3] = 2; A.Values[4] = 1; A.Values[5] = 1;
    A.Values[6] = 2; A.Values[7] = 1; A.Values[8] = 1;
    A.Values[9] = 2;
    A.J[0] = 0; A.J[1] = 1; A.J[2] = 0; A.J[3] = 1; A.J[4] = 2;
    A.J[5] = 1; A.J[6] = 2; A.J[7] = 3; A.J[8] = 2; A.J[9] = 3;
    A.I[0] = 0; A.I[1] = 2; A.I[2] = 5; A.I[3] = 8; A.I[4] = 10;
        
    // Fill b with data.
    b.Values[0] = 1; b.Values[1] = 1; b.Values[2] = 1;
    b.Values[3] = 1;

    // Fill x with data. (initial guess)
    x_0.Values[0] = 1; x_0.Values[1] = 1; x_0.Values[2] = 1;
    x_0.Values[3] = 1;

    // Copy x_0 to x;
    GPU_Dense x = x_0.clone();

    // Initialize solver data and fill with addresses.
    SOLVER_data Axb;
    Axb.A_g = &A;
    Axb.b_g = &b;
    Axb.x_g = &x;
    Axb.n = A.Size[0];
    Axb.nnz = A.Size[2];

    // Specify AMGX configuration.
    AMGX_Mode mode = AMGX_mode_dDDI;
    
    // Start the solving procedure.
    SOLVER_AmgX AMGX(config_spec, mode, Axb);

    // Check for errors.
    if ( abs(x.Values[0] - 0.4) > tol ||
         abs(x.Values[1] - 0.2) > tol ||
         abs(x.Values[2] - 0.2) > tol ||
         abs(x.Values[3] - 0.4) > tol ) {
         pass = 0; }

    // Return the result.
    return pass;

}

int GPU_f_AMGX_test ( std::string config_spec ) {

    // Check.
    int pass = 1;
    float tol = 0.0001;

    // Initialize problem data.
    GPU_Sparse_f A(4,4,10);
    GPU_Dense_f  b(1,4);
    GPU_Dense_f  x_0(1,4);

    // Fill A with data.
    A.Values[0] = 2; A.Values[1] = 1; A.Values[2] = 1;
    A.Values[3] = 2; A.Values[4] = 1; A.Values[5] = 1;
    A.Values[6] = 2; A.Values[7] = 1; A.Values[8] = 1;
    A.Values[9] = 2;
    A.J[0] = 0; A.J[1] = 1; A.J[2] = 0; A.J[3] = 1; A.J[4] = 2;
    A.J[5] = 1; A.J[6] = 2; A.J[7] = 3; A.J[8] = 2; A.J[9] = 3;
    A.I[0] = 0; A.I[1] = 2; A.I[2] = 5; A.I[3] = 8; A.I[4] = 10;
    
    // Fill b with data.
    b.Values[0] = 1; b.Values[1] = 1; b.Values[2] = 1;
    b.Values[3] = 1;

    // Fill x with data. (initial guess)
    x_0.Values[0] = 1; x_0.Values[1] = 1; x_0.Values[2] = 1;
    x_0.Values[3] = 1;

    // Copy x_0 to x;
    GPU_Dense_f x = x_0.clone();

    // Initialize solver data and fill with addresses.
    SOLVER_data Axb;
    Axb.A_gf = &A;
    Axb.b_gf = &b;
    Axb.x_gf = &x;
    Axb.n = A.Size[0];
    Axb.nnz = A.Size[2];

    // Specify AMGX configuration.
    AMGX_Mode mode = AMGX_mode_dFFI;
    
    // Start the solving procedure.
    SOLVER_AmgX AMGX(config_spec, mode, Axb);

    // Check for errors.
    if ( abs(x.Values[0] - 0.4) > tol ||
         abs(x.Values[1] - 0.2) > tol ||
         abs(x.Values[2] - 0.2) > tol ||
         abs(x.Values[3] - 0.4) > tol ) {
         pass = 0; }

    // Return the result.
    return pass;

}

int main (  ) {

    std::string config_spec = "config_version=2, \
                               solver(fgmres)=FGMRES, \
                               fgmres:tolerance=0.01, \
                               fgmres:max_iters=40, \
                               fgmres:gmres_n_restart=10, \
                               fgmres:preconditioner=NOSOLVER";

    int pass1 = CPU_AMGX_test(config_spec);
    int pass2 = CPU_f_AMGX_test(config_spec);
    int pass3 = GPU_AMGX_test(config_spec);
    int pass4 = GPU_f_AMGX_test(config_spec);
    
    int pass_global = 1;

    if (pass1 == 0) { pass_global = 0; std::cout << "Error in: CPU_AMGX_test" << std::endl; }
    if (pass2 == 0) { pass_global = 0; std::cout << "Error in: CPU_f_AMGX_test" << std::endl; }
    if (pass3 == 0) { pass_global = 0; std::cout << "Error in: GPU_AMGX_test" << std::endl; }
    if (pass4 == 0) { pass_global = 0; std::cout << "Error in: GPU_f_AMGX_test" << std::endl; }
        
    if (pass_global == 1) { std::cout << "PASSED" << std::endl; } else { std::cout << "FAILED" << std::endl; }

    return 0;

}