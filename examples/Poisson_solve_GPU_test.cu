#include "hip/hip_runtime.h"
#include "../include/CMaT.h"

int main (  ) {

    // Specify AMGX configuration.
    std::string config_spec = "config_version=2, \
                               solver(fgmres)=FGMRES, \
                               fgmres:tolerance=0.00000001, \
                               fgmres:max_iters=2000, \
                               fgmres:gmres_n_restart=40, \
                               fgmres:preconditioner=NOSOLVER, \
                               fgmres:monitor_residual=1";

    // Open the MAT file.
    matfile_load mat_file_load("./tests/MAT_FILES","Poisson_10000.mat");

    // Load the data (Ar=b).
    GPU_Sparse A(mat_file_load,"A");
    GPU_Dense  b(mat_file_load,"b");
    GPU_Dense  r(mat_file_load,"x");

    // Solve Ax=b.
    tic();
    GPU_Dense x = A.solve(b,config_spec);
    toc();

    // Subtract the r from x.
    GPU_Dense b_x = A.mv(x);
    GPU_Dense res = b_x.add(b.scp(-1));

    // Print the result.
    std::cout << "2-norm of the residual is: " << sqrt(res.dot(res)) << std::endl;

    return 0;

}