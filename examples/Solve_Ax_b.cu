#include "hip/hip_runtime.h"
#include "../include/CMaT.h"
#include <fstream>
#include <ctime>
#include <regex>
#include <stdlib.h>

// Provide filename for log file.
std::string get_output_file_name ( std::string name ) {
    
    // Time and date information.
    std::time_t now = std::time(0);
    tm * ltm = localtime(&now);

    // Date string.
    std::string year  = std::to_string(1900 + ltm->tm_year);
    std::string month = std::to_string(1 + ltm->tm_mon);
    std::string day   = std::to_string(ltm->tm_mday);

    // Time string.
    std::string hour = std::to_string(ltm->tm_hour);
    std::string min  = std::to_string(ltm->tm_min);
    std::string sec  = std::to_string(ltm->tm_sec);

    // year/month/day_hour:min:sec
    std::string time_str =  year+"-"+month+"-"+day+"_"+hour+":"+min+":"+sec;
    
    // Return the name of the output file. 
    return "log/"+name+"_"+time_str+".log";

}

// Create and open log file (overwrite).
std::ofstream create_log ( std::string name, std::string matfile,
                           std::string conffile, std::string backend ) {

    // Create 'log' directory.
    system("if [ ! -d \"log\" ]; then mkdir log; fi");

    // Output file.
    std::ofstream log;
    log.open(get_output_file_name(name));

    // Input parameters
    log << name << " " << matfile << " " << conffile 
        << " " << backend << std::endl << std::endl;

    // Return the log file stream.
    return log;

}

// Input argument processing.
std::string in_arg_check ( int argc, char* argv[] ) {

    // Initialize output.
    std::string check = argv[3];

    if ( argc != 4 ) {

        // Tell the user how to run the program.
        std::cout << "Usage: " << argv[0] 
                  << " MAT_file config_file GPU/CPU" 
                  << std::endl;

        // Error.
        return check = "error";
                    
    }

    // Backend check.
    if ( not((check == "GPU")||(check == "CPU")) ) {

        // Message to user.
        std::cout << "Current backend: " << argv[3] << std::endl;
        std::cout << "Backend should be: 'GPU' or 'CPU'" << std::endl;

        // Error.
        return check = "error";

    }

    // Manipulate the first input argument.
    if ( std::regex_match(argv[0], std::regex("(./)(.*)")) ) {

        // Remove './' from input argument.
        std::string check = argv[0];
        check.erase(0,2);

    }

    // Return the result.
    return check;

}

int main ( int argc, char* argv[] ) {

    // Iput argument processing.
    std::string name = in_arg_check(argc,argv);
    if (name == "error") { return 0; }
    std::string matfile = argv[1], conffile = argv[2], backend = argv[3];
                
    // Log file.
    std::ofstream log = create_log(name, matfile, conffile, backend);
    log.close();
       
    // Open the MAT file.
    matfile_load mat_file_load(matfile);

    // Solve using the CPU.
    if ( backend == "CPU" ) {

        // Extract data to host memory.
        CPU_Sparse A(mat_file_load,"A");
        CPU_Dense  b(mat_file_load,"b");

        // Solve Ax=b (and time).
        tic();
        CPU_Dense x = A.solve(b,conffile);
        toc();

        // Calculate the residual.
        CPU_Dense b_x = A.mv(x);
        CPU_Dense res = b_x.add(b.scp(-1));

        // Print the result.
        std::cout << "2-norm of the residual is: " << sqrt(res.dot(res)) << std::endl;

    }
        
    // Solve using the GPU.
    if ( backend == "GPU" ) {
            
        // Extract data to device memory.
        GPU_Sparse A(mat_file_load,"A");
        GPU_Dense  b(mat_file_load,"b");

        // Solve Ax=b (and time).
        tic();
        GPU_Dense x = A.solve(b,conffile);
        toc();

        // Calculate the residual.
        GPU_Dense b_x = A.mv(x);
        GPU_Dense res = b_x.add(b.scp(-1));

        // Print the result.
        std::cout << "2-norm of the residual is: " << sqrt(res.dot(res)) << std::endl;

    }

    // Exit.
    return 0;

}