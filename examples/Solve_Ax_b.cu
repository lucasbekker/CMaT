#include "hip/hip_runtime.h"
#include "../include/CMaT.h"
#include <fstream>
#include <ctime>
#include <regex>
#include <stdlib.h>

// Provide filename for log file.
std::string get_output_file_name ( std::string name ) {
    
    // Time and date information.
    std::time_t now = std::time(0);
    tm * ltm = localtime(&now);

    // Date string.
    std::string year  = std::to_string(1900 + ltm->tm_year);
    std::string month = std::to_string(1 + ltm->tm_mon);
    std::string day   = std::to_string(ltm->tm_mday);

    // Time string.
    std::string hour = std::to_string(ltm->tm_hour);
    std::string min  = std::to_string(ltm->tm_min);
    std::string sec  = std::to_string(ltm->tm_sec);

    // year-month-day_hour:min:sec
    std::string time_str =  year+"-"+month+"-"+day+"_"+hour+":"+min+":"+sec;
    
    // Return the name of the output file. 
    return "log/"+name+"_"+time_str+".log";

}

// Create and open log file (overwrite).
std::ofstream create_log ( std::string name, std::string matfile, std::string conffile,
                           std::string backend, std::string & filename ) {

    // Create 'log' directory.
    system("if [ ! -d \"log\" ]; then mkdir log; fi");

    // Output file.
    std::ofstream log;
    filename = get_output_file_name(name);
    log.open(filename);

    // Input parameters
    log << name << " " << matfile << " " << conffile 
        << " " << backend << std::endl << std::endl;

    // Return the log file stream.
    return log;

}

// Input argument processing.
std::string in_arg_check ( int argc, char* argv[] ) {

    // Initialize output.
    std::string check = argv[3];

    if ( argc != 4 ) {

        // Tell the user how to run the program.
        std::cout << "Usage: " << argv[0] 
                  << " MAT_file config_file GPU/CPU" 
                  << std::endl;

        // Error.
        return check = "error";
                    
    }

    // Backend check.
    if ( not((check == "GPU")||(check == "CPU")) ) {

        // Message to user.
        std::cout << "Current backend: " << argv[3] << std::endl;
        std::cout << "Backend should be: 'GPU' or 'CPU'" << std::endl;

        // Error.
        return check = "error";

    }

    // Manipulate the first input argument.
    check = argv[0];

    // Remove './' from input argument.
    if ( std::regex_match(argv[0], std::regex("(./)(.*)")) ) {

        check.erase(0,2);

    }

    // Return the result.
    return check;

}

// Store timings, num_iter and norm of residual in a matlab vector.
void create_mat ( double t1, double t2, double t3,
                  double r_norm, std::string filename ) {

    // Initialize the result vector.
    CPU_Dense result(5,1);

    // Fill the result vector.
    result.Values[0] = t1;
    result.Values[1] = t2;
    result.Values[2] = t3;
    result.Values[3] = r_norm;
    result.Values[4] = (double) num_iter;

    // Create './MAT_FILES/results' directory.
    system("if [ ! -d \"MAT_FILES\" ]; then mkdir MAT_FILES; fi");
    system("if [ ! -d \"MAT_FILES/results\" ]; then mkdir MAT_FILES/results; fi");

    // Construct filename.
    filename.pop_back(); filename.pop_back();
    filename.pop_back(); filename.pop_back();
    filename.erase(0,4);

    // Open MAT file for saving.
    matfile_save mat_file_save("./MAT_FILES/results",filename + ".mat");
    
    // Store the result vector in the MAT file.
    result.save(mat_file_save,"result");    

}

int main ( int argc, char* argv[] ) {

    // Iput argument processing.
    std::string name = in_arg_check(argc,argv);
    if (name == "error") { return 0; }
    std::string matfile = argv[1], conffile = argv[2], backend = argv[3];
                
    // Open log file.
    std::string filename;
    std::ofstream log = create_log(name, matfile, conffile, backend, filename);
       
    // Open the MAT file.
    matfile_load mat_file_load(matfile);

    // Timings and residual.
    double t1, t2, t3, r_norm;

    // Solve using the CPU.
    if ( backend == "CPU" ) {

        // Extract data to host memory (and time).
        tic();
        CPU_Sparse A(mat_file_load,"A");
        CPU_Dense  b(mat_file_load,"b");
        t1 = toc_s();
        log << "Loading and transposing took: " << t1 << " seconds" << std::endl;

        // Solve Ax=b (and time).
        tic();
        CPU_Dense x = A.solve(b,conffile);
        t2 = toc_s();
        log << "Solving Ax=b took: " << t2 << " seconds" << std::endl;

        // Calculate the residual.
        tic();
        CPU_Dense b_x = A.mv(x);
        CPU_Dense res = b_x.add(b.scp(-1));
        r_norm = sqrt(res.dot(res));
        t3 = toc_s();
        log << "Getting the residual norm took: " << t3 << " seconds" << std::endl;

        // Print the result.
        log << std::endl << "2-norm of the residual is: " << r_norm << std::endl;
        log << "Number of iterations: " << num_iter << std::endl;

    }
        
    // Solve using the GPU.
    if ( backend == "GPU" ) {
            
        tic();
        GPU_Sparse A(mat_file_load,"A");
        GPU_Dense  b(mat_file_load,"b");
        t1 = toc_s();
        log << "Loading and transposing took: " << t1 << " seconds" << std::endl;

        // Solve Ax=b (and time).
        tic();
        GPU_Dense x = A.solve(b,conffile);
        t2 = toc_s();
        log << "Solving Ax=b took: " << t2 << " seconds" << std::endl;

        // Calculate the residual.
        tic();
        GPU_Dense b_x = A.mv(x);
        GPU_Dense res = b_x.add(b.scp(-1));
        r_norm = sqrt(res.dot(res));
        t3 = toc_s();
        log << "Getting the residual norm took: " << t3 << " seconds" << std::endl;

        // Print the result.
        log << std::endl << "2-norm of the residual is: " << r_norm << std::endl;
        log << "Number of iterations: " << num_iter << std::endl;

    }

    create_mat(t1, t2, t3, r_norm, filename);

    // Close log file.
    log.close();

    // Exit.
    return 0;

}