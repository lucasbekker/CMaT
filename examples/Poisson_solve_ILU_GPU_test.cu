#include "hip/hip_runtime.h"
#include "../include/CMaT.h"

int main (  ) {

    // Specify AMGX configuration.
    std::string config_spec = "config_version=2, \
                               solver(main)=FGMRES, \
                               main:max_iters=10000, \
                               main:convergence=RELATIVE_MAX, \
                               main:gmres_n_restart=100, \
                               main:tolerance=0.00000001, \
                               main:monitor_residual=1, \
                               main:preconditioner(ilu)=NOSOLVER, \
                               main:print_solve_stats=1, \
                               determinism_flag=1";

    // Open the MAT file.
    matfile_load mat_file_load("./tests/MAT_FILES","jacobi_daan_19801.mat");

    // Load the data (Ar=b).
    GPU_Sparse A(mat_file_load,"A");
    GPU_Dense  b(mat_file_load,"b");
    GPU_Dense  r(mat_file_load,"x");

    // Solve Ax=b.
    tic();
    GPU_Dense x = A.solve(b,config_spec);
    toc();

    // Subtract the r from x.
    GPU_Dense b_x = A.mv(x);
    GPU_Dense res = b_x.add(b.scp(-1));

    // Print the result.
    std::cout << "2-norm of the residual is: " << sqrt(res.dot(res)) << std::endl;

    return 0;

}