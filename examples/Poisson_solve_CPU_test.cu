#include "hip/hip_runtime.h"
#include "../include/CMaT.h"

int main (  ) {

    /*// Specify AMGX configuration.
    std::string config_spec = "config_version=2, \
                               solver(fgmres)=FGMRES, \
                               fgmres:tolerance=0.00000001, \
                               fgmres:max_iters=2000, \
                               fgmres:gmres_n_restart=40, \
                               fgmres:preconditioner=NOSOLVER, \
                               fgmres:monitor_residual=1";*/

    std::string config_spec = "./config/GMRES.json";

    // Open the MAT file.
    matfile_load mat_file_load("./tests/MAT_FILES","Poisson_10000.mat");

    // Load the data (Ar=b).
    CPU_Sparse A(mat_file_load,"A");
    CPU_Dense  b(mat_file_load,"b");
    CPU_Dense  r(mat_file_load,"x");

    // Solve Ax=b.
    tic();
    CPU_Dense x = A.solve(b,config_spec);
    toc();

    // Subtract the r from x.
    CPU_Dense b_x = A.mv(x);
    CPU_Dense res = b_x.add(b.scp(-1));

    // Print the result.
    std::cout << "2-norm of the residual is: " << sqrt(res.dot(res)) << std::endl;

    return 0;

}